#include "hip/hip_runtime.h"
// #include <iostream>
// #include <fstream>
// #include <sstream>
// #include <vector>
// #include <numeric>
// #include <random>
// #include <cmath>
// #include <chrono>
// #include <vector>
// #include <unordered_map>
// #include <hip/hip_runtime.h>
// #include "common.h"

// #define TOTAL_AGENT 6897434
// #define coefficient1 0.00000928
// // #define coefficient1 0.00928
// #define coefficient2 0.00843


// void initialize_basic_variables(const char* agent_txt_name, const char* neighbor_txt_name, int*simulation_agents_number_pointer, float*scale_pointer, long* neighbors_number_pointer ){
//     std::ifstream file(agent_txt_name);
//     if (file.is_open()) {
//         file >> (*simulation_agents_number_pointer);
//         *scale_pointer = (*simulation_agents_number_pointer) / TOTAL_AGENT;
//         std::cout << "simulation_agents_number: "<< (*simulation_agents_number_pointer) <<  "\n";
//         file.close();
//     }
//     std::ifstream file2(neighbor_txt_name);
//     if (file2.is_open()) {
//         file2 >> (*neighbors_number_pointer);
//         std::cout << "neighbor_number: "<< (*neighbors_number_pointer) <<  "\n";
//         file2.close();
//     }

// }

// void initialize_agents(const char* txt_name , Agent* agents){
//     int simulation_agents_number;
//     std::ifstream file("agents.txt");
//     if (file.is_open()) {
//         file >> simulation_agents_number;
//         int zip,num_neigh;
//         float inc;
//         for ( int i =0; i < simulation_agents_number ; i++){
//             file >> zip >> inc >> num_neigh;
//             agents[i] = Agent(zip, inc, num_neigh);
//         }
//         file.close();
//     }
// }

// void initialize_neighbors(const char* txt_name, int* neighbors_array){
//     std::ifstream file(txt_name);
//     long neighbors_number;
//     if (file.is_open()) {
//         file >> neighbors_number;
//         int neighbor;
//         for ( int i =0; i < neighbors_number ; i++){
//             file >> neighbor;
//             neighbors_array[i] = neighbor;
//         }
//     }
// }

// void initialize_neighbor_prefix(const char* txt_name, int* neighbors_prefix_sum_pointer){
//     std::ifstream file(txt_name);
//     int simulation_agents_number;
//     if (file.is_open()) {
//         file >> simulation_agents_number;
//         int temp;
//         for ( int i =0; i < simulation_agents_number ; i++){
//             file >> temp;
//             neighbors_prefix_sum_pointer[i] = temp;
//         }
//     }
// }



// void initialize_number_neighbors_each_agent(const char* txt_name, int* number_neighbors_each_agent, int* agents_max_degree_pointer){
//     std::ifstream file(txt_name);
//     int simulation_agents_number;
//     if (file.is_open()) {
//         file >> simulation_agents_number;
//         int temp;
//         for ( int i =0; i < simulation_agents_number ; i++){
//             file >> temp;
//             number_neighbors_each_agent[i] = temp;
//             if (temp > (*agents_max_degree_pointer)){
//                 (*agents_max_degree_pointer) = temp;
//             }
//         }
//     }
// }


// int step(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
//          int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption, float* random_number_array){
//     // std::mt19937 gen(seed); 
//     // std::uniform_real_distribution<> dis(0.0, 1.0);
//     agents[0].affected = 1;
//     // std::cout<< simulation_agents_number_s<<std::endl;
//     for (int j = 0 ; j < simulation_agents_number ; j++){
//         //float might not be proper
//         double random_num = random_number_array[current_step*simulation_agents_number + j];
//         double threshold;
//         if (agents[j].affected == 1){
//             continue;
//         }
//         int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
//         float agent_income = agents[j].income;
//         threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
//         // std::cout << threshold << random_num << std::endl;
//         if (threshold > random_num){
//             // std::cout << "enter" << std::endl;
//             current_number_adoption += 1;
//             agents[j].affected = 1;
//             for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
//                 agents[neighbors_array[k]].number_of_neighbors_affected += 1;
//             }
//         }
//     }
//     // std::cout << current_number_adoption<<std::endl;
//     return current_number_adoption;
// }



// int step_later_update(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
//          int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption, float* radom_number_array){
//     // std::mt19937 gen(seed); 
//     // std::uniform_real_distribution<> dis(0.0, 1.0);
//     std::vector<int> affected_this_step;
//     agents[0].affected = 1;
//     // std::cout<< simulation_agents_number_s<<std::endl;
//     for (int j = 0 ; j < simulation_agents_number ; j++){
//         //float might not be proper
//         double random_num = radom_number_array[current_step*simulation_agents_number + j];
//         double threshold;
//         if (agents[j].affected == 1){
//             continue;
//         }
//         int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
//         float agent_income = agents[j].income;
//         threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
//         // std::cout << threshold << random_num << std::endl;
//         if (threshold > random_num){
//             // std::cout << "enter" << std::endl;
//             current_number_adoption += 1;
//             agents[j].affected = 1;
//             affected_this_step.push_back(j);
//             // for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
//             //     agents[neighbors_array[k]].number_of_neighbors_affected += 1;
//             // }
//         }
//     }

//     for (std::vector<int>::size_type j = 0 ; j < affected_this_step.size() ; j++){
//         int current_value = affected_this_step[j];
//         for(int k = neighbors_prefix_sum[current_value]; k < neighbors_prefix_sum[current_value] + number_neighbors_each_agent[current_value] ; k++){
//             agents[neighbors_array[k]].number_of_neighbors_affected += 1;
//         }
//     }

//     // std::cout << current_number_adoption<<std::endl;
//     return current_number_adoption;
// }


// // int step_all(int total_simulation_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
// //          int* number_adoption, int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption,int seed=12){
// //     std::mt19937 gen(seed); 
// //     std::uniform_real_distribution<> dis(0.0, 1.0);
// //     // std::cout<< simulation_agents_number_s<<std::endl;
// //     current_number_adoption = 0;
// //     for(int i =0; i <total_simulation_step;i++ ){

// //     for (int j = 0 ; j < simulation_agents_number ; j++){
// //         //float might not be proper
// //         double random_num = dis(gen);
// //         double threshold;
// //         if (agents[j].affected == 1){
// //             continue;
// //         }
// //         int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
// //         float agent_income = agents[j].income;
// //         threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
// //         // std::cout << threshold << random_num << std::endl;
// //         if (threshold > random_num){
// //             current_number_adoption += 1;
// //             agents[j].affected = 1;
// //             for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
// //                 agents[neighbors_array[k]].number_of_neighbors_affected += 1;
// //             }
// //         }
// //     }
// //     std::cout << current_number_adoption<<std::endl;
// //     }
// //     return current_number_adoption;
// // }

// // void transition(int total_simulation_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
// //              int* number_adoption,int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int seed=12){
// //     int current_number_adoption = 0;
// //     for(int i =0; i < total_simulation_step;i++){
// //         current_number_adoption = step(simulation_agents_number, agents,agents_max_degree, 
// //         neighbors_prefix_sum,neighbors_array,number_neighbors_each_agent,current_number_adoption,seed);
// //         number_adoption[i] = current_number_adoption;
// //         std::cout << number_adoption[i] << std::endl;
// //     }

// // }


// // int step(int simulation_agents_number, Agent* agents, int agents_max_degree, 
// //          int* number_adoption, int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption,int seed=12){
// //     std::mt19937 gen(seed); 
// //     std::uniform_real_distribution<> dis(0.0, 1.0);
// //     for (int j = 0 ; j < simulation_agents_number ; j++){
// //         //float might not be proper
// //         double random_num = dis(gen);
// //         double threshold;
// //         if (agents[j].affected == 1){
// //             continue;
// //         }
// //         int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
// //         float agent_income = agents[j].income;
// //         threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
// //         if (threshold > random_num){
// //             // std::cout << "enter" << std::endl;
// //             current_number_adoption += 1;
// //             agents[j].affected = 1;
// //             for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
// //                 agents[neighbors_array[k]].number_of_neighbors_affected += 1;
// //             }
// //         }
// //     }
// //     return current_number_adoption;
// // }


// int main() {
//     int total_simulation_step = 700;
//     std::vector<int> number_adoption(total_simulation_step);



//     // Agent* agents = new Agent[];
//     // std::vector<Agent> agents;
//     int simulation_agents_number;

//     std::vector<int> neighbors_array;
//     long neighbors_number;
//     std::vector<int> neighbors_prefix_sum;
//     std::vector<int> number_neighbors_each_agent;
//     int agents_max_degree = 0;
//     float scale;
//     initialize_basic_variables("agents.txt", "neighbors.txt",&simulation_agents_number, &scale, &neighbors_number );
//     Agent* agents = new Agent[simulation_agents_number];
//     std::mt19937 gen(12); 
//     std::uniform_real_distribution<> dis(0.0, 1.0);
//     std::vector<float> random_number_array(total_simulation_step*simulation_agents_number);
//     for(int i =0; i < total_simulation_step*simulation_agents_number;i++){
//         random_number_array[i] = dis(gen);
//     }
//     std::cout<<"finish generate"<<std::endl;
//     // agents.resize(simulation_agents_number);
//     neighbors_prefix_sum.resize(simulation_agents_number);
//     number_neighbors_each_agent.resize(simulation_agents_number);
//     neighbors_array.resize(neighbors_number);

//     initialize_agents("agents.txt", agents);
//     initialize_neighbors("neighbors.txt", neighbors_array.data());
//     initialize_neighbor_prefix("neighbor_prefix.txt", neighbors_prefix_sum.data());

//     initialize_number_neighbors_each_agent("number_neighbors_each_agent.txt", number_neighbors_each_agent.data(), &agents_max_degree);

//     int current_number_adoption = 0;

//     // step_all(total_simulation_step,simulation_agents_number, agents,agents_max_degree, 
//     //      number_adoption.data(), neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data(),current_number_adoption);

//     for(int i =0 ; i< total_simulation_step;i++){
//         current_number_adoption = step(i,simulation_agents_number, agents,agents_max_degree, 
//          neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data(),current_number_adoption,random_number_array.data());
//         number_adoption[i] = current_number_adoption;
//         std::cout << number_adoption[i] << std::endl;
//     }

//     // transition(total_simulation_step,simulation_agents_number, agents,agents_max_degree, 
//     //      number_adoption.data(), neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data());

//     // step( simulation_agents_number, agents.data(),agents_max_degree, 
//     //      number_adoption.data(), neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data());
    
//     // for (int i=0; i < simulation_agents_number; i++){
//     //   if(agents[i].affected){
//     //     std::cout << agents[i].zipcode<<std::endl;
//     //   }
//     // }
//     return 0;
// }

















#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <numeric>
#include <random>
#include <cmath>
#include <chrono>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include "common.h"

#define TOTAL_AGENT 6897434
#define coefficient1 0.00000928
#define coefficient2 0.00843
#define NUM_THREADS 256

void initialize_basic_variables(const char* agent_txt_name, const char* neighbor_txt_name, int*simulation_agents_number_pointer, float*scale_pointer, long* neighbors_number_pointer ){
    std::ifstream file(agent_txt_name);
    if (file.is_open()) {
        file >> (*simulation_agents_number_pointer);
        *scale_pointer = (*simulation_agents_number_pointer) / TOTAL_AGENT;
        std::cout << "simulation_agents_number: "<< (*simulation_agents_number_pointer) <<  "\n";
        file.close();
    }
    std::ifstream file2(neighbor_txt_name);
    if (file2.is_open()) {
        file2 >> (*neighbors_number_pointer);
        std::cout << "neighbor_number: "<< (*neighbors_number_pointer) <<  "\n";
        file2.close();
    }

}

void initialize_agents(const char* txt_name , Agent* agents){
    int simulation_agents_number;
    std::ifstream file("agents.txt");
    if (file.is_open()) {
        file >> simulation_agents_number;
        int zip,num_neigh;
        float inc;
        for ( int i =0; i < simulation_agents_number ; i++){
            file >> zip >> inc >> num_neigh;
            agents[i] = Agent(zip, inc, num_neigh);
        }
        file.close();
    }
}

void initialize_neighbors(const char* txt_name, int* neighbors_array){
    std::ifstream file(txt_name);
    long neighbors_number;
    if (file.is_open()) {
        file >> neighbors_number;
        int neighbor;
        for ( int i =0; i < neighbors_number ; i++){
            file >> neighbor;
            neighbors_array[i] = neighbor;
        }
    }
}

void initialize_neighbor_prefix(const char* txt_name, int* neighbors_prefix_sum_pointer){
    std::ifstream file(txt_name);
    int simulation_agents_number;
    if (file.is_open()) {
        file >> simulation_agents_number;
        int temp;
        for ( int i =0; i < simulation_agents_number ; i++){
            file >> temp;
            neighbors_prefix_sum_pointer[i] = temp;
        }
    }
}



void initialize_number_neighbors_each_agent(const char* txt_name, int* number_neighbors_each_agent, int* agents_max_degree_pointer){
    std::ifstream file(txt_name);
    int simulation_agents_number;
    if (file.is_open()) {
        file >> simulation_agents_number;
        int temp;
        for ( int i =0; i < simulation_agents_number ; i++){
            file >> temp;
            number_neighbors_each_agent[i] = temp;
            if (temp > (*agents_max_degree_pointer)){
                (*agents_max_degree_pointer) = temp;
            }
        }
    }
}


int step(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
         int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption, float* radom_number_array){
    // std::mt19937 gen(seed); 
    // std::uniform_real_distribution<> dis(0.0, 1.0);
    
    agents[0].affected = 1;
    // std::cout<< simulation_agents_number_s<<std::endl;
    for (int j = 0 ; j < simulation_agents_number ; j++){
        //float might not be proper
        double random_num = radom_number_array[current_step*simulation_agents_number + j];
        double threshold;
        if (agents[j].affected == 1){
            continue;
        }
        int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
        float agent_income = agents[j].income;
        threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
        // std::cout << threshold << random_num << std::endl;
        if (threshold > random_num){
            // std::cout << "enter" << std::endl;
            current_number_adoption += 1;
            agents[j].affected = 1;
            for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
                agents[neighbors_array[k]].number_of_neighbors_affected += 1;
            }
        }
    }
    // std::cout << current_number_adoption<<std::endl;
    return current_number_adoption;
}


int step_later_update(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
         int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption, float* radom_number_array){
    // std::mt19937 gen(seed); 
    // std::uniform_real_distribution<> dis(0.0, 1.0);
    std::vector<int> affected_this_step;
    agents[0].affected = 1;
    // std::cout<< simulation_agents_number_s<<std::endl;
    for (int j = 0 ; j < simulation_agents_number ; j++){
        //float might not be proper
        double random_num = radom_number_array[current_step*simulation_agents_number + j];
        double threshold;
        if (agents[j].affected == 1){
            continue;
        }
        int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
        float agent_income = agents[j].income;
        threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
        // std::cout << threshold << random_num << std::endl;
        if (threshold > random_num){
            // std::cout << "enter" << std::endl;
            current_number_adoption += 1;
            agents[j].affected = 1;
            affected_this_step.push_back(j);
            // for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
            //     agents[neighbors_array[k]].number_of_neighbors_affected += 1;
            // }
        }
    }

    for (std::vector<int>::size_type j = 0 ; j < affected_this_step.size() ; j++){
        int current_value = affected_this_step[j];
        // std::cout << "agent:" << current_value << "# neighbor" << number_neighbors_each_agent[current_value] << std::endl;
        for(int k = neighbors_prefix_sum[current_value]; k < neighbors_prefix_sum[current_value] + number_neighbors_each_agent[current_value] ; k++){
            agents[neighbors_array[k]].number_of_neighbors_affected += 1;
        }
    }

    // std::cout << current_number_adoption<<std::endl;
    return current_number_adoption;
}

int find_arg_idx(int argc, char** argv, const char* option) {
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], option) == 0) {
            return i;
        }
    }
    return -1;
}


int find_int_arg(int argc, char** argv, const char* option, int default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return std::stoi(argv[iplace + 1]);
    }

    return default_value;
}


float find_float_arg(int argc, char** argv, const char* option, float default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return std::stof(argv[iplace + 1]);
    }

    return default_value;
}


int main(int argc, char**argv) {

    int total_simulation_step = 700;
    total_simulation_step = find_int_arg(argc, argv,"-steps", 1000);
    std::vector<int> number_adoption(total_simulation_step);
    int simulation_agents_number;

    long neighbors_number;
    int agents_max_degree = 0;
    float scale;
    initialize_basic_variables("agents.txt", "neighbors.txt",&simulation_agents_number, &scale, &neighbors_number );
    Agent* agents = new Agent[simulation_agents_number];
    int* neighbors_array = new int[neighbors_number];
    int* neighbors_prefix_sum = new int[simulation_agents_number];
    int* number_neighbors_each_agent = new int[simulation_agents_number];
    std::mt19937 gen(12); 
    std::uniform_real_distribution<> dis(0.0, 1.0);
    // std::vector<float> random_number_array(total_simulation_step*simulation_agents_number);
    float* random_number_array = new float[total_simulation_step*simulation_agents_number];
    for(int i =0; i < total_simulation_step*simulation_agents_number;i++){
        random_number_array[i] = dis(gen);
    }
    int block_number_agents = ceil(simulation_agents_number/NUM_THREADS) + 1;
    int block_number_neighbors=ceil(neighbors_number/NUM_THREADS) + 1;
    int current_number_adoption = 0;
    float* agent_threshold_array = new float[simulation_agents_number];
    for(int i =0; i < simulation_agents_number;i++){
        agent_threshold_array[i] = 0;
    }


    int* current_adoption_array = new int[total_simulation_step];
    for(int i =0; i < total_simulation_step;i++){
        current_adoption_array[i] = 0;
    }
    std::cout<<"finish generate"<<std::endl;

    initialize_agents("agents.txt", agents);
    initialize_neighbors("neighbors.txt", neighbors_array);
    initialize_neighbor_prefix("neighbor_prefix.txt", neighbors_prefix_sum);

    initialize_number_neighbors_each_agent("number_neighbors_each_agent.txt", number_neighbors_each_agent, &agents_max_degree);


    auto start = std::chrono::high_resolution_clock::now();
    for(int i =0 ; i< total_simulation_step;i++){
        current_number_adoption = step_later_update(i,simulation_agents_number, agents,agents_max_degree, 
         neighbors_prefix_sum,neighbors_array,number_neighbors_each_agent,current_number_adoption,random_number_array);
        number_adoption[i] = current_number_adoption;
        std::cout << number_adoption[i] << std::endl;
    }



    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  
    std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

    delete[] agents;
    delete[] neighbors_prefix_sum;
    delete[] number_neighbors_each_agent;
    delete[] random_number_array;
    delete[] neighbors_array;
    return 0;
}

