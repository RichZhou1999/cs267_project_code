#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <numeric>
#include <random>
#include <cmath>
#include <chrono>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include "common.h"

#define TOTAL_AGENT 6897434
#define coefficient1 0.00000928
// #define coefficient1 0.00928
#define coefficient2 0.00843
#define NUM_THREADS 256


Agent* d_agents;
int* d_neighbors_prefix_sum;
int* d_neighbors_array;
int* d_number_neighbors_each_agent;
float* d_random_number_array;
float* d_agent_threshold_array;
int* d_current_adoption_array;
int* d_adoption_history_array;

void initialize_basic_variables(const char* agent_txt_name, const char* neighbor_txt_name, int*simulation_agents_number_pointer, float*scale_pointer, long* neighbors_number_pointer ){
    std::ifstream file(agent_txt_name);
    if (file.is_open()) {
        file >> (*simulation_agents_number_pointer);
        *scale_pointer = (*simulation_agents_number_pointer) / TOTAL_AGENT;
        std::cout << "simulation_agents_number: "<< (*simulation_agents_number_pointer) <<  "\n";
        file.close();
    }
    std::ifstream file2(neighbor_txt_name);
    if (file2.is_open()) {
        file2 >> (*neighbors_number_pointer);
        std::cout << "neighbor_number: "<< (*neighbors_number_pointer) <<  "\n";
        file2.close();
    }

}

void initialize_agents(const char* txt_name , Agent* agents){
    int simulation_agents_number;
    std::ifstream file("agents.txt");
    if (file.is_open()) {
        file >> simulation_agents_number;
        int zip,num_neigh;
        float inc;
        for ( int i =0; i < simulation_agents_number ; i++){
            file >> zip >> inc >> num_neigh;
            agents[i] = Agent(zip, inc, num_neigh);
        }
        file.close();
    }
}

void initialize_neighbors(const char* txt_name, int* neighbors_array){
    std::ifstream file(txt_name);
    long neighbors_number;
    if (file.is_open()) {
        file >> neighbors_number;
        int neighbor;
        for ( int i =0; i < neighbors_number ; i++){
            file >> neighbor;
            neighbors_array[i] = neighbor;
        }
    }
}

void initialize_neighbor_prefix(const char* txt_name, int* neighbors_prefix_sum_pointer){
    std::ifstream file(txt_name);
    int simulation_agents_number;
    if (file.is_open()) {
        file >> simulation_agents_number;
        int temp;
        for ( int i =0; i < simulation_agents_number ; i++){
            file >> temp;
            neighbors_prefix_sum_pointer[i] = temp;
        }
    }
}



void initialize_number_neighbors_each_agent(const char* txt_name, int* number_neighbors_each_agent, int* agents_max_degree_pointer){
    std::ifstream file(txt_name);
    int simulation_agents_number;
    if (file.is_open()) {
        file >> simulation_agents_number;
        int temp;
        for ( int i =0; i < simulation_agents_number ; i++){
            file >> temp;
            number_neighbors_each_agent[i] = temp;
            if (temp > (*agents_max_degree_pointer)){
                (*agents_max_degree_pointer) = temp;
            }
        }
    }
}


int step(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
         int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption, float* radom_number_array){
    // std::mt19937 gen(seed); 
    // std::uniform_real_distribution<> dis(0.0, 1.0);
    
    agents[0].affected = 1;
    // std::cout<< simulation_agents_number_s<<std::endl;
    for (int j = 0 ; j < simulation_agents_number ; j++){
        //float might not be proper
        double random_num = radom_number_array[current_step*simulation_agents_number + j];
        double threshold;
        if (agents[j].affected == 1){
            continue;
        }
        int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
        float agent_income = agents[j].income;
        threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
        // std::cout << threshold << random_num << std::endl;
        if (threshold > random_num){
            // std::cout << "enter" << std::endl;
            current_number_adoption += 1;
            agents[j].affected = 1;
            for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
                agents[neighbors_array[k]].number_of_neighbors_affected += 1;
            }
        }
    }
    // std::cout << current_number_adoption<<std::endl;
    return current_number_adoption;
}


int step_later_update(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
         int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption, float* radom_number_array){
    // std::mt19937 gen(seed); 
    // std::uniform_real_distribution<> dis(0.0, 1.0);
    std::vector<int> affected_this_step;
    agents[0].affected = 1;
    // std::cout<< simulation_agents_number_s<<std::endl;
    for (int j = 0 ; j < simulation_agents_number ; j++){
        //float might not be proper
        double random_num = radom_number_array[current_step*simulation_agents_number + j];
        double threshold;
        if (agents[j].affected == 1){
            continue;
        }
        int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
        float agent_income = agents[j].income;
        threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
        // std::cout << threshold << random_num << std::endl;
        if (threshold > random_num){
            // std::cout << "enter" << std::endl;
            current_number_adoption += 1;
            agents[j].affected = 1;
            affected_this_step.push_back(j);
            // for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
            //     agents[neighbors_array[k]].number_of_neighbors_affected += 1;
            // }
        }
    }

    for (std::vector<int>::size_type j = 0 ; j < affected_this_step.size() ; j++){
        int current_value = affected_this_step[j];
        for(int k = neighbors_prefix_sum[current_value]; k < neighbors_prefix_sum[current_value] + number_neighbors_each_agent[current_value] ; k++){
            agents[neighbors_array[k]].number_of_neighbors_affected += 1;
        }
    }

    // std::cout << current_number_adoption<<std::endl;
    return current_number_adoption;
}


// int step_all(int total_simulation_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
//          int* number_adoption, int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption,int seed=12){
//     std::mt19937 gen(seed); 
//     std::uniform_real_distribution<> dis(0.0, 1.0);
//     // std::cout<< simulation_agents_number_s<<std::endl;
//     current_number_adoption = 0;
//     for(int i =0; i <total_simulation_step;i++ ){

//     for (int j = 0 ; j < simulation_agents_number ; j++){
//         //float might not be proper
//         double random_num = dis(gen);
//         double threshold;
//         if (agents[j].affected == 1){
//             continue;
//         }
//         int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
//         float agent_income = agents[j].income;
//         threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
//         // std::cout << threshold << random_num << std::endl;
//         if (threshold > random_num){
//             current_number_adoption += 1;
//             agents[j].affected = 1;
//             for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
//                 agents[neighbors_array[k]].number_of_neighbors_affected += 1;
//             }
//         }
//     }
//     std::cout << current_number_adoption<<std::endl;
//     }
//     return current_number_adoption;
// }


// int step(int simulation_agents_number, Agent* agents, int agents_max_degree, 
//          int* number_adoption, int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int current_number_adoption,int seed=12){
//     std::mt19937 gen(seed); 
//     std::uniform_real_distribution<> dis(0.0, 1.0);
//     for (int j = 0 ; j < simulation_agents_number ; j++){
//         //float might not be proper
//         double random_num = dis(gen);
//         double threshold;
//         if (agents[j].affected == 1){
//             continue;
//         }
//         int agent_current_num_neighbor = agents[j].number_of_neighbors_affected;
//         float agent_income = agents[j].income;
//         threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
//         if (threshold > random_num){
//             // std::cout << "enter" << std::endl;
//             current_number_adoption += 1;
//             agents[j].affected = 1;
//             for(int k = neighbors_prefix_sum[j]; k < neighbors_prefix_sum[j] + number_neighbors_each_agent[j] ; k++){
//                 agents[neighbors_array[k]].number_of_neighbors_affected += 1;
//             }
//         }
//     }
//     return current_number_adoption;
// }

__global__ void copy_agent_objects(Agent* des, Agent* source, int num_items){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_items){
        return;
    }

    des[tid] = source[tid];
}

__global__ void copy_int_objects(int* des, int* source, int num_items){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_items){
        return;
    }

    des[tid] = source[tid];
}

__global__ void copy_float_objects(float* des, float* source, int num_items){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_items){
        return;
    }

    des[tid] = source[tid];
}


// __global__ void calculate_threshold(int current_step,int simulation_agents_number, Agent* agents, int agents_max_degree,float* random_number_array,
//                                     float* agent_threshold_array)
//     {
//         int tid = threadIdx.x + blockIdx.x * blockDim.x;
//         if (tid >= simulation_agents_number)
//         {
//         return;
//         }
//         Agent* agent = &agents[tid];
//         float threshold;
//         // int agent_current_num_neighbor = agent->number_of_neighbors_affected;
//         float agent_income = agent->income;
//         // threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
//         // agent_threshold_array[tid] = threshold;
//     }


__global__ void calculate_threshold(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
    float* agent_threshold_array){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= simulation_agents_number)
    {
    return;
    }
    Agent* agent = &agents[tid];
    if (agent->affected == 1){
        return;
    }
    float threshold;
    int agent_current_num_neighbor = agent->number_of_neighbors_affected;
    float agent_income = agent->income;
    threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
    agent_threshold_array[tid] = threshold;
    }


__global__ void step_gpu(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
    int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int* current_number_adoption, float* random_number_array){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= simulation_agents_number)
    {
    return;
    }
    Agent* agent = &agents[tid];
    if (agent->affected == 1){
        return;
    }
    float random_num = random_number_array[current_step*simulation_agents_number + tid];
    double threshold;
    int agent_current_num_neighbor = agent->number_of_neighbors_affected;
    float agent_income = agent->income;
    threshold = coefficient1 * agent_income + coefficient2 * agent_current_num_neighbor/agents_max_degree;
    // std::cout << threshold << random_num << std::endl;
    if (threshold > random_num){
        // std::cout << "enter" << std::endl;
        // printf("enter \n");
        // printf("threshold %f random_number:%f",threshold,random_num);
        // current_number_adoption += 1;
        agent->affected = 1;
        atomicAdd(current_number_adoption,1);
        for(int k = neighbors_prefix_sum[tid]; k < neighbors_prefix_sum[tid] + number_neighbors_each_agent[tid] ; k++){
            atomicAdd(&((&agents[neighbors_array[k]])->number_of_neighbors_affected),1);
            // agents[neighbors_array[k]].number_of_neighbors_affected += 1;
        }
    }
    }



__global__ void update_adoption(int current_step, int simulation_agents_number, Agent* agents, int agents_max_degree, 
    int* neighbors_prefix_sum, int*neighbors_array,int* number_neighbors_each_agent,int* current_number_adoption, float* random_number_array, float* agent_threshold_array, int* adoption_history_array){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= simulation_agents_number)
    {
    return;
    }
    Agent* agent = &agents[tid];
    if (agent->affected == 1){
        return;
    }
    float random_num = random_number_array[current_step*simulation_agents_number + tid];
    float threshold = agent_threshold_array[tid];
    if (threshold > random_num){
        agent->affected = 1;
        int temp;
        temp = atomicAdd(current_number_adoption,1);
        adoption_history_array[temp] = tid;
        // for(int k = neighbors_prefix_sum[tid]; k < neighbors_prefix_sum[tid] + number_neighbors_each_agent[tid] ; k++){
        //     atomicAdd(&((&agents[neighbors_array[k]])->number_of_neighbors_affected),1);
        // }
    }
    }

__global__ void update_affection(int agent_index,int number,Agent* agents,int* neighbors_prefix_sum, int* neighbors_array){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= number)
    {
    return;
    }
    // Agent* agent = &agents[agent_index];
    int start_position = neighbors_prefix_sum[agent_index];
    (&agents[neighbors_array[start_position+tid]])->number_of_neighbors_affected += 1;
}

__global__ void record_current_number_adoption(int* current_number_adoption, int* current_adoption_array, int step){
    
    current_adoption_array[step] = *current_number_adoption;
    printf("%d \n", *current_number_adoption);
}

__global__ void print_random_number_array(float* random_number_array,int number){
    for(int i =0; i <number;i++ ){
        printf("%f \n",random_number_array[i]);
    }
}
int find_arg_idx(int argc, char** argv, const char* option) {
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], option) == 0) {
            return i;
        }
    }
    return -1;
}


int find_int_arg(int argc, char** argv, const char* option, int default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return std::stoi(argv[iplace + 1]);
    }

    return default_value;
}


float find_float_arg(int argc, char** argv, const char* option, float default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return std::stof(argv[iplace + 1]);
    }

    return default_value;
}

int main(int argc, char**argv) {
    int total_simulation_step = 700;
    total_simulation_step = find_int_arg(argc, argv,"-steps", 1000);
    std::vector<int> number_adoption(total_simulation_step);



    int simulation_agents_number;

    long neighbors_number;
    int agents_max_degree = 0;
    float scale;
    initialize_basic_variables("agents.txt", "neighbors.txt",&simulation_agents_number, &scale, &neighbors_number );
    Agent* agents = new Agent[simulation_agents_number];
    int* neighbors_array = new int[neighbors_number];
    int* neighbors_prefix_sum = new int[simulation_agents_number];
    int* number_neighbors_each_agent = new int[simulation_agents_number];
    int* adoption_history_array = new int[simulation_agents_number];
    std::mt19937 gen(12); 
    std::uniform_real_distribution<> dis(0.0, 1.0);
    // std::vector<float> random_number_array(total_simulation_step*simulation_agents_number);
    float* random_number_array = new float[total_simulation_step*simulation_agents_number];
    for(int i =0; i < total_simulation_step*simulation_agents_number;i++){
        random_number_array[i] = dis(gen);
    }
    int block_number_agents = ceil(simulation_agents_number/NUM_THREADS) + 1;
    int block_number_neighbors=ceil(neighbors_number/NUM_THREADS) + 1;
    
    int current_number_adoption = 0;
    int* d_current_number_adoption;
    float* agent_threshold_array = new float[simulation_agents_number];
    for(int i =0; i < simulation_agents_number;i++){
        agent_threshold_array[i] = 0;
    }


    int* current_adoption_array = new int[total_simulation_step];
    for(int i =0; i < total_simulation_step;i++){
        current_adoption_array[i] = 0;
    }
    std::cout<<"finish generate"<<std::endl;

    initialize_agents("agents.txt", agents);
    initialize_neighbors("neighbors.txt", neighbors_array);
    initialize_neighbor_prefix("neighbor_prefix.txt", neighbors_prefix_sum);

    initialize_number_neighbors_each_agent("number_neighbors_each_agent.txt", number_neighbors_each_agent, &agents_max_degree);



    hipMalloc((void**)&d_random_number_array, simulation_agents_number*total_simulation_step* sizeof(float));
    hipMemcpy(d_random_number_array, random_number_array,simulation_agents_number*total_simulation_step* sizeof(float),hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();



    hipMalloc((void**)&d_current_number_adoption, sizeof(int));
    hipMemcpy(d_current_number_adoption, &current_number_adoption, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_agents, simulation_agents_number* sizeof(Agent));
    // copy_agent_objects<<<block_number_agents,NUM_THREADS >>>(d_agents,agents,simulation_agents_number);
    hipMemcpy(d_agents, agents, simulation_agents_number* sizeof(Agent), hipMemcpyHostToDevice);

    // hipDeviceSynchronize();
    // hipError_t cudaerr = hipDeviceSynchronize();
    // if (cudaerr != hipSuccess)
    //     printf("kernel launch failed with error \"%s\".\n",hipGetErrorString(cudaerr));
    hipMalloc((void**)&d_neighbors_prefix_sum, simulation_agents_number* sizeof(int));
    hipMemcpy(d_neighbors_prefix_sum, neighbors_prefix_sum, simulation_agents_number* sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_number_neighbors_each_agent, simulation_agents_number* sizeof(int));
    hipMemcpy(d_number_neighbors_each_agent, number_neighbors_each_agent, simulation_agents_number* sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_neighbors_array, neighbors_number* sizeof(int));
    hipMemcpy(d_neighbors_array, neighbors_array, neighbors_number* sizeof(int), hipMemcpyHostToDevice);

    // hipMalloc((void**)&d_random_number_array, simulation_agents_number*total_simulation_step* sizeof(float));
    // hipMemcpy(d_random_number_array, random_number_array,simulation_agents_number*total_simulation_step* sizeof(float),hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_current_adoption_array, total_simulation_step* sizeof(int));
    hipMemcpy(d_current_adoption_array, current_adoption_array, total_simulation_step* sizeof(int), hipMemcpyHostToDevice);


    
    hipMalloc((void**)&d_agent_threshold_array, simulation_agents_number* sizeof(float));
    hipMemcpy(d_agent_threshold_array, agent_threshold_array,simulation_agents_number* sizeof(float),hipMemcpyHostToDevice);

    hipMalloc((void**)&d_adoption_history_array, simulation_agents_number* sizeof(int));
    hipDeviceSynchronize();
    // for(int i=0; i < simulation_agents_number;i++){
    //     std::cout <<  random_number_array[i] << std::endl;
    // }
    // print_random_number_array<<<1,1>>>(d_random_number_array,simulation_agents_number);
    // hipError_t cudaerr = hipDeviceSynchronize();
    // if (cudaerr != hipSuccess)
    //     printf("kernel launch failed with error \"%s\".\n",hipGetErrorString(cudaerr));


    // step_all(total_simulation_step,simulation_agents_number, agents,agents_max_degree, 
    //      number_adoption.data(), neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data(),current_number_adoption);

    // auto start = std::chrono::high_resolution_clock::now();
    // for(int i =0 ; i< total_simulation_step;i++){
    //     current_number_adoption = step_later_update(i,simulation_agents_number, agents,agents_max_degree, 
    //     neighbors_prefix_sum,neighbors_array,number_neighbors_each_agent,current_number_adoption,random_number_array);
    //     number_adoption[i] = current_number_adoption;
    //     std::cout << number_adoption[i] << std::endl;
    // }


    int previous_number_adoption = 0;
    for(int i =0 ; i< total_simulation_step;i++){
        // calculate_threshold<<<block_number_agents,NUM_THREADS>>>(i,simulation_agents_number, agents, agents_max_degree,d_random_number_array,
        //                             d_agent_threshold_array);
        calculate_threshold<<<block_number_agents,NUM_THREADS>>>(i,simulation_agents_number, d_agents,agents_max_degree, 
         d_agent_threshold_array);
        // hipDeviceSynchronize();
        update_adoption<<<block_number_agents,NUM_THREADS>>>(i,simulation_agents_number, d_agents,agents_max_degree, 
         d_neighbors_prefix_sum,d_neighbors_array,d_number_neighbors_each_agent,d_current_number_adoption,d_random_number_array,d_agent_threshold_array,d_adoption_history_array);
        record_current_number_adoption<<<1,1>>>(d_current_number_adoption,d_current_adoption_array,i);
        hipMemcpy(&current_number_adoption, d_current_number_adoption, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(adoption_history_array + previous_number_adoption, d_adoption_history_array+previous_number_adoption, sizeof(int)*(current_number_adoption-previous_number_adoption), hipMemcpyDeviceToHost);
        // hipMemcpy(adoption_history_array, d_adoption_history_array, sizeof(int)*(simulation_agents_number), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        current_adoption_array[i] = current_number_adoption;
        for(int j =previous_number_adoption; j < current_number_adoption;j++){
            int number_neighbors_one_agent = number_neighbors_each_agent[adoption_history_array[j]];
            int temp_block_num = ceil(number_neighbors_one_agent/NUM_THREADS) + 1;
            // std::cout <<"agent:" <<adoption_history_array[j] << "# neighbor"<< number_neighbors_one_agent<<std::endl;
            update_affection<<<temp_block_num,NUM_THREADS>>>(adoption_history_array[j],number_neighbors_one_agent,d_agents,d_neighbors_prefix_sum, d_neighbors_array);
        }
        hipDeviceSynchronize();
        previous_number_adoption = current_number_adoption;
    }

    // hipError_t cudaerr = hipDeviceSynchronize();
    // if (cudaerr != hipSuccess)
    //     printf("kernel launch failed with error \"%s\".\n",hipGetErrorString(cudaerr));




    // for(int i =0 ; i< total_simulation_step;i++){
    //     step_gpu<<<block_number_agents,NUM_THREADS>>>(i,simulation_agents_number, d_agents,agents_max_degree, 
    //      d_neighbors_prefix_sum,d_neighbors_array,d_number_neighbors_each_agent,d_current_number_adoption,d_random_number_array);
    //     // step_gpu<<<block_number_agents,NUM_THREADS>>>(i,simulation_agents_number, d_agents,agents_max_degree, 
    //     //  d_neighbors_prefix_sum,d_neighbors_array,d_number_neighbors_each_agent,d_current_number_adoption,d_random_number_array);
    //     hipDeviceSynchronize();
    //     record_current_number_adoption<<<1,1>>>(d_current_number_adoption,d_current_adoption_array,i);
    // }

    // transition(total_simulation_step,simulation_agents_number, agents,agents_max_degree, 
    //      number_adoption.data(), neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data());

    // step( simulation_agents_number, agents.data(),agents_max_degree, 
    //      number_adoption.data(), neighbors_prefix_sum.data(),neighbors_array.data(),number_neighbors_each_agent.data());
    
    // for (int i=0; i < simulation_agents_number; i++){
    //   if(agents[i].affected){
    //     std::cout << agents[i].zipcode<<std::endl;
    //   }
    // }
    

    // for(int i =0 ; i< total_simulation_step;i++){
    //     current_number_adoption = step(i,simulation_agents_number, agents,agents_max_degree, 
    //      neighbors_prefix_sum,neighbors_array,number_neighbors_each_agent,current_number_adoption,random_number_array);
    //     number_adoption[i] = current_number_adoption;
    //     std::cout << number_adoption[i] << std::endl;
    // }


    // for(int i =0 ; i< total_simulation_step;i++){
    //     current_number_adoption = step(i,simulation_agents_number, agents,agents_max_degree, 
    //      neighbors_prefix_sum,neighbors_array,number_neighbors_each_agent,current_number_adoption,random_number_array);
    //     number_adoption[i] = current_number_adoption;
    //     std::cout << number_adoption[i] << std::endl;
    // }

    // hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  
    std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;


    delete[] agents;
    delete[] neighbors_prefix_sum;
    delete[] number_neighbors_each_agent;
    delete[] random_number_array;
    delete[] neighbors_array;
    hipFree(d_neighbors_array);
    hipFree(d_agents);
    hipFree(d_neighbors_prefix_sum);
    hipFree(d_number_neighbors_each_agent);
    hipFree(d_random_number_array);
    return 0;
}

